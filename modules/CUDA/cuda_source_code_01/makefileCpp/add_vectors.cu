
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void add_arrays_gpu( float *in1, float *in2, float *out, int Ntot)
{
       int idx=blockIdx.x*blockDim.x+threadIdx.x;
       if ( idx <Ntot )
       out[idx]=in1[idx]+in2[idx]+threadIdx.x;
}

extern
int add_vectors() {
	/* pointers to host memory */
	float *a, *b, *c;
	/* pointers to device memory */
	float *a_d, *b_d, *c_d;
	int N=18;
	int i;

	/* Allocate arrays a, b and c on host*/
	a = (float*) malloc(N*sizeof(float));
	b = (float*) malloc(N*sizeof(float));
	c = (float*) malloc(N*sizeof(float));

	/* Allocate arrays a_d, b_d and c_d on device*/
	hipMalloc ((void **) &a_d, sizeof(float)*N);
	hipMalloc ((void **) &b_d, sizeof(float)*N);
	hipMalloc ((void **) &c_d, sizeof(float)*N);
	
	/* Initialize arrays a and b */
	for (i=0; i<N; i++) {
		a[i]= (float) i;
		b[i]=-(float) i;
	}


	/* Copy data from host memory to device memory */
	hipMemcpy(a_d, a, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, sizeof(float)*N, hipMemcpyHostToDevice);

	/* Compute the execution configuration */
	int block_size=8;
	dim3 dimBlock(block_size);
	dim3 dimGrid ( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );

	/* Add arrays a and b, store result in c */
	add_arrays_gpu<<<dimGrid,dimBlock>>>(a_d, b_d, c_d, N);

	/* Copy data from deveice memory to host memory */
	hipMemcpy(c, c_d, sizeof(float)*N, hipMemcpyDeviceToHost);

	/* Print c */
	for (i=0; i<N; i++)
		printf(" c[%d]=%f\n",i,c[i]);

	/* Free the memory */
	free(a); free(b); free(c);
	hipFree(a_d); hipFree(b_d);hipFree(c_d);

	return 0;
}
