
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void add_arrays_gpu( float *in1, float *in2, float *out, int Ntot) {
       int idx=blockIdx.x*blockDim.x+threadIdx.x;
       if ( idx <Ntot )
       out[idx]=in1[idx]+in2[idx];
}

int main() {
	// pointers to host memory
	float *a, *b, *c;
	// pointers to device memory
	float *a_d, *b_d, *c_d;
	int N=18;
	int i;

	// Allocate arrays a, b and c on host
	a = (float*) malloc(N*sizeof(float));
	b = (float*) malloc(N*sizeof(float));
	c = (float*) malloc(N*sizeof(float));

	// Allocate arrays a_d, b_d and c_d on device
	hipMalloc ((void **) &a_d, sizeof(float)*N);
	hipMalloc ((void **) &b_d, sizeof(float)*N);
	hipMalloc ((void **) &c_d, sizeof(float)*N);

	// Initialize arrays a and b
	for (i=0; i<N; i++) {
		a[i]= (float) 2*i;
		b[i]=-(float) i;
	}

	// Copy data from host memory to device memory
	hipMemcpy(a_d, a, sizeof(float)*N, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, sizeof(float)*N, hipMemcpyHostToDevice);

	// Compute the execution configuration
	int block_size=8;
	dim3 dimBlock(block_size);
	dim3 dimGrid ( (N/dimBlock.x) + (!(N%dimBlock.x)?0:1) );

	// Add arrays a and b, store result in c
	add_arrays_gpu<<<dimGrid,dimBlock>>>(a_d, b_d, c_d, N);

	// Copy data from deveice memory to host memory
	hipMemcpy(c, c_d, sizeof(float)*N, hipMemcpyDeviceToHost);

	// Print c
	printf("addVectors will generate two vectors, move them to the global memory, and add them together in the GPU\n");
	for (i=0; i<N; i++) {
		printf(" a[%2d](%10f) + b[%2d](%10f) = c[%2d](%10f)\n",i,a[i],i,b[i],i,c[i]);
	}

	// Free the memory
	free(a); free(b); free(c);
	hipFree(a_d); hipFree(b_d);hipFree(c_d);
}
